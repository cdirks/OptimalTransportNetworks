#include "hip/hip_runtime.h"
#include <cudaInterface.h>
#include <cudaVector.h>

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////

__global__  void jacobi_kernel(float* device_A, float* device_f, float* device_u_new, float* device_u_old, int nnu) {
  /* get global thread id */
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  /* check if id corresponds to a variable */
  if (i<nnu) {
    float temp=device_f[i];
    for (int j=0;j < i  ;++j)
      temp -= device_A[i*nnu+j]*device_u_old[j];
    for (int j=i+1;j<nnu;++j)
      temp -= device_A[i*nnu+j]*device_u_old[j];
    device_u_new[i]=temp/device_A[i*nnu+i];
  }
}

void cudaJacobiSolver ( float *pMatrix, float *pRhs, float *pSolution, const int Length ) {
  /* Allocation for Memory on device */
  cuda::GPUVector<float> device_A ( Length*Length );
  cuda::GPUVector<float> device_f ( Length );
  cuda::GPUVector<float> device_u_new ( Length );
  cuda::GPUVector<float> device_u_old ( Length );

  /* Copy from host to device */
  device_A.copyFromHost ( pMatrix );
  device_f.copyFromHost ( pRhs );
  device_u_new.setZero();
  device_u_old.setZero();

  /* use one thread for every variable
  organize line of blocks with 512 threads each in a grid
  until number of variables reached */
  int block_size = 512;
  dim3 dimBlock(block_size,1,1);
  int dimenx = Length/block_size;
  if (dimenx*block_size < Length)
    dimenx++;
  dim3 dimGrid(dimenx,1,1);

  for ( int i = 0; i < 1000; ++i ) {
    /* start kernel code on device */
    jacobi_kernel<<<dimGrid, dimBlock>>>(device_A.getDataPointer(), device_f.getDataPointer(), device_u_new.getDataPointer(), device_u_old.getDataPointer(), Length);
    device_u_old = device_u_new;
  }

  /* copy back to host */
  device_u_new.copyToHost( pSolution );
}
