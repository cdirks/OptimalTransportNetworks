#include "hip/hip_runtime.h"
#include <cudaInterface.h>
#include <cudaVector.h>

////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////

//divergenz
__device__ float div ( float* p1, float* p2, int i, int j, int nx, int ny ) {
  float div = 0;

  if ( i == 0 )
    div += p1[ cuda::getGlobalIndex ( i, j, nx ) ];
  if( i == nx-1 )
    div += -p1[ cuda::getGlobalIndex ( i-1, j, nx ) ];
  if ( 0 < i && i < nx-1 )
    div += p1[ cuda::getGlobalIndex ( i, j, nx ) ] - p1[ cuda::getGlobalIndex ( i-1, j, nx ) ];

  if ( j == 0 )
    div += p2[ cuda::getGlobalIndex ( i, j, nx ) ];
  if ( j == ny-1 )
    div += -p2[ cuda::getGlobalIndex ( i, j-1, nx ) ];
  if ( 0 < j && j < ny-1 )
    div += p2[ cuda::getGlobalIndex ( i, j, nx ) ]-p2[ cuda::getGlobalIndex ( i, j-1, nx ) ];

  return div;
}

__device__ float dXFD2D ( float* image, int x, int y, int nx, int ny ) {
  return image[ cuda::getGlobalIndex ( x + 1, y, nx ) ] - image[ cuda::getGlobalIndex ( x, y, nx ) ];
}

__device__ float dYFD2D ( float* image, int x, int y, int nx, int ny ) {
  return image[ cuda::getGlobalIndex ( x, y + 1, nx ) ] - image[ cuda::getGlobalIndex ( x, y, nx ) ];
}

////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////

class TVChambolleHelper {
public:
  static __device__ float func ( float* p1, float* p2, float *image, float *, int x, int y, int nx, int ny ) {
    return image[ cuda::getGlobalIndex ( x, y, nx ) ] - div ( p1, p2, x, y, nx, ny );
  };
};

class MSSegHelper {
public:
  static __device__ float func ( float* p1, float* p2, float *twiceIndicator2OverLambda, float *twiceIndicator1Plus2, int x, int y, int nx, int ny ) {
    const int globalIndex = cuda::getGlobalIndex ( x, y, nx );
    return ( twiceIndicator2OverLambda[globalIndex] - div ( p1, p2, x, y, nx, ny ) ) / twiceIndicator1Plus2[globalIndex];
  };
};

template <typename T>
__global__ void update_duals ( float* p1, float* p2, float* p1New, float* p2New,
                               float* in1, float* in2, float timestep,
                               int nx, int ny) {
  // calculate point in colormap discretization
  unsigned int ix = blockDim.x*blockIdx.x + threadIdx.x;
  unsigned int iy = blockDim.y*blockIdx.y + threadIdx.y;

  // for bigger thread numbers there isn't a corresponding point in colormap discretization
  if ( ( ix >= nx ) || ( iy >= ny ) )
    return;

  const int tx = threadIdx.x;
  const int ty = threadIdx.y;
  const int globalIdx = cuda::getGlobalIndex ( ix, iy, nx );

  // calculate ( image - div p ) and store it in shared memory, so that all threads can access it.
  // The amound of memory necessary for this is specified in the execution configuration of update_duals.
  extern __shared__ float imageMinusDivP[];
  imageMinusDivP[ cuda::getGlobalIndex ( tx, ty, blockDim.x + 1 ) ] =  T::func ( p1, p2, in1, in2, ix, iy, nx, ny );
  if ( tx == blockDim.x-1 )
    imageMinusDivP[ cuda::getGlobalIndex ( tx + 1, ty, blockDim.x + 1 ) ] = ( ix < nx-1 ) ? T::func ( p1, p2, in1, in2, ix + 1, iy, nx, ny ) : 0;
  if ( ty == blockDim.y-1 )
    imageMinusDivP[ cuda::getGlobalIndex ( tx, ty + 1, blockDim.x + 1 ) ] = ( iy < ny-1 ) ? T::func ( p1, p2, in1, in2, ix, iy + 1, nx, ny ) : 0;
  __syncthreads();

  //-\nabla colormap + \nabla ind *div p + ind*\nabla div p
  const float buffer1 = ( ix < nx-1 ) ? - dXFD2D ( imageMinusDivP, tx, ty, blockDim.x + 1, blockDim.y + 1 ) : 0;
  const float buffer2 = ( iy < ny-1 ) ? - dYFD2D ( imageMinusDivP, tx, ty, blockDim.x + 1, blockDim.y + 1 ) : 0;

  const float norm = sqrt(buffer1*buffer1+buffer2*buffer2);

  //update p1, p2
  p1New[globalIdx] = ( p1[globalIdx] + timestep*buffer1 ) / ( 1 + timestep*norm);
  p2New[globalIdx] = ( p2[globalIdx] + timestep*buffer2 ) / ( 1 + timestep*norm);
}

////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////

template <typename T>
__global__ void constructImageFromDuals ( float* in1, float* in2, float* p1, float* p2,
                                          float lambda, int nx, int ny) {
  // calculate point in colormap discretization
  unsigned int ix = blockDim.x*blockIdx.x + threadIdx.x;
  unsigned int iy = blockDim.y*blockIdx.y + threadIdx.y;

  // for bigger thread numbers there isn't a corresponding point in colormap discretization
  if ( ( ix >= nx ) || ( iy >= ny ) )
    return;

  in1[ cuda::getGlobalIndex ( ix, iy, nx ) ] = lambda * T::func( p1, p2, in1, in2, ix, iy, nx, ny );
}

////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////

template <typename T>
void ChambolleTVAlgo ( cuda::GPUVector<float> &DevIn1, cuda::GPUVector<float> &DevIn2, int nx, int ny, float lambda, float tol, float timeStep, int maxIter ) {
  // Size of a thread block, should be optimized for the GPU the code is run on.
  // The size of 16x16 seems to work well for a GeForce 8800 GT.
  const int blockSizeX = 16;
  const int blockSizeY = 16;

  ////////////////////////////////////////////////////////////////////////////////
  // compute number of Blocks & Grid
  cuda::GridBlockConfig2D gridConf(  blockSizeX, blockSizeY, nx, ny );

  cuda::GPUMultiVector<float> dev_p ( 2, nx*ny ); 
  dev_p.setZero();
  cuda::GPUMultiVector<float> dev_pNew ( 2, nx*ny );
  dev_pNew.setZero();

  ////////////////////////////////////////////////////////////////////////////////

  hipblasStatus_t stat = cublasInit();
  if (stat != HIPBLAS_STATUS_SUCCESS) {
    printf ("Failed to initialize the CUBLAS library\n" );
  }

  cuda::GPUMultiVector<float> errorMVec ( 2, nx*ny );
  errorMVec.setZero();

  float error = tol + 1;
  //calculate dual variables p1,p2 through fix point iteration scheme
  for ( int i = 1; i < maxIter; ++i ) {
    //calculate p^n+1= (p^n + timestep*buffer) / (1+timestep*norm)
    // with     buffer =  \nabla (div p^n - image / lambda)
    // and      norm = ||buffer||_2
    update_duals<T><<<gridConf.dimGrid, gridConf.dimBlock, (gridConf.dimBlock.x+1) * (gridConf.dimBlock.y+1) * sizeof (float) >>>(dev_p[0].getDataPointer(), dev_p[1].getDataPointer(), dev_pNew[0].getDataPointer(), dev_pNew[1].getDataPointer(), DevIn1.getDataPointer(), DevIn2.getDataPointer(), timeStep, nx, ny);
    CUT_CHECK_ERROR("ERROR: dual Kernel Failed !");
    dev_p = dev_pNew;
    if ( i % 100 == 0 ) {
      //save old variables
      errorMVec = dev_p;
    }
    if ( i % 100 == 1 ) {
      //difference of old and new
      errorMVec -= dev_p;
      error = errorMVec[0].norm() + errorMVec[1].norm();
    }

    if ( abs(error) < tol ) {
      break;
    }
  }

  //reconstruct colormap from duals
  constructImageFromDuals<T><<<gridConf.dimGrid, gridConf.dimBlock>>>(DevIn1.getDataPointer(), DevIn2.getDataPointer(), dev_p[0].getDataPointer(), dev_p[1].getDataPointer(), lambda, nx, ny);
  CUT_CHECK_ERROR("ERROR: construct Kernel Failed !");
}

void tv_smooth_gpu(float* image, int nx, int ny, float lambda, float tol, float timeStep, int maxIter) {

  ////////////////////////////////////////////////////////////////////////////////
  //allocation & init

  cuda::GPUVector<float> dev_image ( nx*ny );
  dev_image.copyFromHost ( image );
  dev_image *= 1.f / lambda;

  ChambolleTVAlgo<TVChambolleHelper>( dev_image, dev_image, nx, ny, lambda, tol, timeStep, maxIter);

  ////////////////////////////////////////////////////////////////////////////////
  // copy back to host
  dev_image.copyToHost ( image );
}

/**
 * Result is stored in indicator1.
 *
 * \author Berkels
 */
void MSSegGPU(float* indicator1, float* indicator2, int nx, int ny, float lambda, float tol, float timeStep, int maxIter) {
  ////////////////////////////////////////////////////////////////////////////////
  //allocation & init
  const float lambdaHDependent = lambda * ( ( nx > ny ) ? nx - 1 : ny - 1 );

  cuda::GPUVector<float> twiceIndicator1Plus2 ( nx*ny );
  twiceIndicator1Plus2.copyFromHost ( indicator1 );
  cuda::GPUVector<float> twiceIndicator2OverLambda ( nx*ny );
  twiceIndicator2OverLambda.copyFromHost ( indicator2 );
  twiceIndicator1Plus2 += twiceIndicator2OverLambda;
  twiceIndicator1Plus2 *= 2.f;
  twiceIndicator2OverLambda *= 2.f / lambdaHDependent;

  ChambolleTVAlgo<MSSegHelper>( twiceIndicator2OverLambda, twiceIndicator1Plus2, nx, ny, lambdaHDependent, tol, timeStep, maxIter);

  ////////////////////////////////////////////////////////////////////////////////
  // copy back to host
  twiceIndicator2OverLambda.copyToHost ( indicator1 );
}
