#include "hip/hip_runtime.h"
#include <cudaInterface.h>
#include <cudaVector.h>

namespace cuda {

template <typename DataType>
GPUMemoryBlock<DataType>::GPUMemoryBlock ( int Size )
  : _pData ( cuda::allocate<DataType> ( Size ) ),
    _size ( Size ) {
}

template <typename DataType>
GPUMemoryBlock<DataType>::~GPUMemoryBlock () {
  cuda::deallocate ( _pData );
}

template <typename DataType>
void GPUMemoryBlock<DataType>::setZero () {
  cuda::memset ( _pData, 0, _size );
}

template <typename DataType>
void GPUMemoryBlock<DataType>::copyFromHost ( const DataType * const PHostMem ) {
  cuda::memcpyHostToGPU ( _pData, PHostMem, _size );
}

template <typename DataType>
void GPUMemoryBlock<DataType>::copyToHost ( DataType *PHostMem ) const {
  cuda::memcpyGPUToHost ( PHostMem, _pData, _size );
}

template class GPUMemoryBlock<double>;
template class GPUMemoryBlock<float>;
template class GPUMemoryBlock<int>;

template <typename RealType>
GPUVector<RealType>::GPUVector ( const GPUVector<RealType> &Vec, const int CopyFlag )
  : GPUMemoryBlock<RealType> ( Vec._size ) {
  assert ( ( CopyFlag == 1 ) || ( CopyFlag == 2 ) );

  if ( CopyFlag == 1 )
    *this = Vec;
  else if ( CopyFlag == 2 )
    this->setZero();
  else
    printf ( "Unexpected CopyFlag value specified.\n" );
}

template <typename RealType>
GPUVector<RealType>& GPUVector<RealType>::operator= ( const GPUVector<RealType> &Vec ) {
  assert ( Vec.size() == this->_size );

  // Beware of self-assignment
  if ( this->_pData != Vec._pData )
    cuda::memcpyGPUToGPU ( this->_pData, Vec._pData, this->_size );

  return *this;
}
template <typename RealType>

GPUVector<RealType>& GPUVector<RealType>::operator+= ( const GPUVector<RealType> &Vec ) {
  return addMultiple ( Vec, 1 );
}
template <typename RealType>

GPUVector<RealType>& GPUVector<RealType>::operator-= ( const GPUVector<RealType> &Vec ) {
  return addMultiple ( Vec, -1 );
}

template <typename RealType>
GPUVector<RealType>& GPUVector<RealType>::operator*= ( const RealType Value ) {
  cublasScale<RealType> ( this->_size, Value, this->_pData );
  return *this;
}

template <typename RealType>
RealType GPUVector<RealType>::operator* ( const GPUVector<RealType> &Vec ) const {
  return cublasScalarProduct<RealType> ( this->_size, this->_pData, Vec._pData );
}

template <typename RealType>
RealType GPUVector<RealType>::norm ( ) const {
  return cublasNormSqr<RealType> ( this->_size, this->_pData ); 
}

template <typename RealType>
GPUVector<RealType>& GPUVector<RealType>::addMultiple ( const GPUVector<RealType> &Vec, RealType Factor ) {
  cublasAddMultiple<RealType> ( this->_size, Factor, Vec._pData, this->_pData ); 
  return *this;
}

template class GPUVector<double>;
template class GPUVector<float>;

void initLibCublas ( ) {
  hipblasStatus_t stat = cublasInit(); 
  if ( stat != HIPBLAS_STATUS_SUCCESS ) {
    printf ( "Failed to initialize the CUBLAS library\n" );
  }
}

}
