#include <cudaInterface.h>

namespace cuda {

void printCurrentDevice ( ) {
  hipDeviceProp_t deviceProp;
  int devID = 0;
  hipGetDevice ( &devID );
  CUDA_SAFE_CALL( hipGetDeviceProperties(&deviceProp, devID) );
  printf("Current CUDA device is %s (%d MB total global memory).\n", deviceProp.name, ( deviceProp.totalGlobalMem ) / ( 1024 * 1024 ) );
}

template <>
void cublasScale ( const int Size, const float Value, float *PData ) {
  hipblasSscal ( Size, Value, PData, 1 );
}

template <>
void cublasScale ( const int Size, const double Value, double *PData ) {
  hipblasDscal ( Size, Value, PData, 1 );
}

template <>
float cublasScalarProduct ( const int Size, const float *PDataA, const float *PDataB ) {
 return hipblasSdot ( Size, PDataA, 1, PDataB, 1 );
}

template <>
double cublasScalarProduct ( const int Size, const double *PDataA, const double *PDataB ) {
 return hipblasDdot ( Size, PDataA, 1, PDataB, 1 );
}

template <>
float cublasNormSqr ( const int Size, const float *PData ) {
  return hipblasSnrm2 ( Size, PData, 1 ); 
}

template <>
double cublasNormSqr ( const int Size, const double *PData ) {
  return hipblasDnrm2 ( Size, PData, 1 ); 
}

template <>
void cublasAddMultiple ( const int Size, const float Factor, const float *PDataArg, float *PDataDest ) {
  hipblasSaxpy ( Size, Factor, PDataArg, 1, PDataDest, 1); 
}

template <>
void cublasAddMultiple ( const int Size, const double Factor, const double *PDataArg, double *PDataDest ) {
  hipblasDaxpy ( Size, Factor, PDataArg, 1, PDataDest, 1); 
}

}
