#include "hip/hip_runtime.h"
#include <cudaInterface.h>
#include <cudaMatrix.h>

namespace cuda {

template <typename RealType>
GPUFullMatrix<RealType>::GPUFullMatrix ( const int NumRows, const int NumCols )
  : _data ( NumRows*NumCols ),
    _numRows ( NumRows ),
    _numCols ( NumCols ) {
}

template <typename RealType>
void GPUFullMatrix<RealType>::setZero () {
  _data.setZero();
}

template <typename RealType>
void GPUFullMatrix<RealType>::apply ( const GPUVector<RealType> &Arg, GPUVector<RealType> &Dest ) const {
  hipblasSgemv ( 'n', _numRows, _numCols, 1, _data.getDataPointer(), _numRows, Arg.getDataPointer(), 1, 0, Dest.getDataPointer(), 1);
}

template <typename RealType>
void GPUFullMatrix<RealType>::applyAdd ( const GPUVector<RealType> &Arg, GPUVector<RealType> &Dest ) const {
  hipblasSgemv ( 'n', _numRows, _numCols, 1, _data.getDataPointer(), _numRows, Arg.getDataPointer(), 1, 1, Dest.getDataPointer(), 1);
}

template <typename RealType>
void GPUFullMatrix<RealType>::copyFromHost ( const RealType * const PHostMem, const bool Flip ) {
  if ( Flip == false )
    _data.copyFromHost ( PHostMem );
  else {
    RealType *pBuffer = new RealType [ _numRows * _numCols ];
    for ( int i = 0; i < _numRows; ++i )
      for ( int j = 0; j < _numCols; ++j )
        pBuffer[i + j*_numRows] = PHostMem[i*_numCols + j];
    _data.copyFromHost ( pBuffer );
    delete[] pBuffer;
  }
}

template class GPUFullMatrix<float>;

template <typename RealType>
void GPUDiagonalMatrix<RealType>::apply ( const GPUVector<RealType> &Arg, GPUVector<RealType> &Dest ) const {
  // In order to use cubals to apply our diagonal vector as matrix, we interpret this vector as band matrix
  // with zero sub- and superdiagonals.
  hipblasSgbmv ('n', getNumRows(), getNumCols(), 0, 0, 1, this->getDataPointer(), 1, Arg.getDataPointer(), 1, 0, Dest.getDataPointer(), 1);
}

template class GPUDiagonalMatrix<float>;

/**
 * \author Berkels
 */
template <typename RealType>
__device__  void cache2DArray ( const RealType* Arg, RealType* ArgCached, const int ix, const int iy, const int Width, const int CacheSizeX, const int row ) {

  const int tx = threadIdx.x;
  const int ty = threadIdx.y;

  ArgCached[getGlobalIndex ( tx + 1, ty + 1, CacheSizeX )] = Arg[row];

  if ( tx == blockDim.x-1 )
    ArgCached[ getGlobalIndex ( tx + 2, ty + 1, CacheSizeX ) ] = ( ix < Width - 1 ) ? Arg [ getGlobalIndex ( ix + 1, iy, Width ) ] : 0;
  else if ( tx == 0 )
    ArgCached[ getGlobalIndex ( tx, ty + 1, CacheSizeX ) ] = ( ix > 0 ) ? Arg [ getGlobalIndex ( ix - 1, iy, Width ) ] : 0;
  if ( ty == blockDim.y-1 )
    ArgCached[ getGlobalIndex ( tx + 1, ty + 2, CacheSizeX ) ] = ( iy < Width - 1 ) ? Arg [ getGlobalIndex ( ix, iy + 1, Width ) ] : 0;
  else if ( ty == 0 )
    ArgCached[ getGlobalIndex ( tx + 1, ty, CacheSizeX ) ] = ( iy > 0 ) ? Arg [ getGlobalIndex ( ix, iy - 1, Width ) ] : 0;

  if ( ( tx == 0 ) && ( ty == 0 ) )
    ArgCached[ getGlobalIndex ( tx, ty, CacheSizeX ) ] = ( ( ix > 0 ) && ( iy > 0 ) ) ? Arg [ getGlobalIndex ( ix - 1, iy - 1, Width ) ] : 0;
  else if ( ( tx == 0 ) && ( ty == blockDim.y-1 ) )
    ArgCached[ getGlobalIndex ( tx, ty + 2, CacheSizeX ) ] = ( ( ix > 0 ) && ( iy < Width - 1 ) ) ? Arg [ getGlobalIndex ( ix - 1, iy + 1, Width ) ] : 0;
  else if ( ( tx == blockDim.x-1 ) && ( ty == 0 ) )
    ArgCached[ getGlobalIndex ( tx + 2, ty, CacheSizeX ) ] = ( ( ix < Width - 1 ) && ( iy > 0 ) ) ? Arg [ getGlobalIndex ( ix + 1, iy - 1, Width ) ] : 0;
  else if ( ( tx == blockDim.x-1 ) && ( ty == blockDim.y-1 ) )
    ArgCached[ getGlobalIndex ( tx + 2, ty + 2, CacheSizeX ) ] = ( ( ix < Width - 1 ) && ( iy < Width - 1 ) ) ? Arg [ getGlobalIndex ( ix + 1, iy + 1, Width ) ] : 0;
}

/**
 * \author Berkels
 */
template <typename RealType>
__device__  void applyRowOf2DQuadraticGridMatrix ( const RealType* Rows, const RealType* ArgCached, RealType* Dest, const int CacheSizeX, const int row ) {
  int k = 0;

  const int tx = threadIdx.x;
  const int ty = threadIdx.y;

  Dest[row] = 0;
  for ( int j = -1; j <= 1; ++j ) {
    int Y = ty + 1 + j;
    for ( int i = -1; i <= 1; ++i ) {
      int X = tx + 1 + i;
      Dest[row] += Rows[row*9+k] * ArgCached[ getGlobalIndex ( X, Y, CacheSizeX ) ];
      ++k;
    }
  }
}

/**
 * A kernel to apply cuda::GPU2DQuadraticGridMatrix to a vector while respecting that
 * the vector actually is a 2D array. Assumes to be used with 2D blocks.
 *
 * \author Berkels
 */
__global__  void apply2DQuadraticGridMatrix ( const float* Rows, const float* Arg, float* Dest, const int NumDofs, const int Width ) {
  // calculate point coordinates in the 2D grid
  unsigned int ix = blockDim.x*blockIdx.x + threadIdx.x;
  unsigned int iy = blockDim.y*blockIdx.y + threadIdx.y;

  // for bigger thread numbers there isn't a corresponding grid point
  if ( ( ix >= Width ) || ( iy >= Width ) )
    return;

  const int row = getGlobalIndex ( ix, iy, Width );

  extern __shared__ float argCached[];
  const int cacheSizeX = blockDim.x + 2;

  cache2DArray<float> ( Arg, argCached, ix, iy, Width, cacheSizeX, row );
  __syncthreads();
  applyRowOf2DQuadraticGridMatrix<float> ( Rows, argCached, Dest, cacheSizeX, row );
}

/**
 * Double version of cuda::GPU2DQuadraticGridMatrix. Unfortunately CUDA doesn't seem to allow to
 * templetize the type of shared memory (and doesn't like two versions of a kernel to use the same
 * name for their shared memory, so this code duplication can't be completely prevented.
 *
 * \author Berkels
 */
__global__  void apply2DQuadraticGridMatrix ( const double* Rows, const double* Arg, double* Dest, const int NumDofs, const int Width ) {
// Older architectures don't support double precision.
#if __CUDA_ARCH__ >= 130
  // calculate point coordinates in the 2D grid
  unsigned int ix = blockDim.x*blockIdx.x + threadIdx.x;
  unsigned int iy = blockDim.y*blockIdx.y + threadIdx.y;

  // for bigger thread numbers there isn't a corresponding grid point
  if ( ( ix >= Width ) || ( iy >= Width ) )
    return;

  const int row = getGlobalIndex ( ix, iy, Width );

  extern __shared__ double argCachedDouble[];
  const int cacheSizeX = blockDim.x + 2;

  cache2DArray<double> ( Arg, argCachedDouble, ix, iy, Width, cacheSizeX, row );
  __syncthreads();
  applyRowOf2DQuadraticGridMatrix<double> ( Rows, argCachedDouble, Dest, cacheSizeX, row );
#endif
}

/**
 * Straightforward implementation of a kernel to apply cuda::GPU2DQuadraticGridMatrix to a vector.
 * Assumes to be used with 1D blocks.
 *
 * \author Berkels
 */
__global__  void apply2DQuadraticGridMatrixSimple ( const float* Rows, const float* Arg, float* Dest, const int NumDofs, const int Width ) {
  int row = blockIdx.x * blockDim.x + threadIdx.x;

  if ( row < NumDofs ) {
    Dest[row] = 0;
    int startIndex = row - Width - 1;
    int k = 0;

    int y = row / Width;
    int x = row % Width;

    for ( int j = -1; j <= 1; ++j ) {
      int Y = y + j;
      for ( int i = -1; i <= 1; ++i ) {
        int X = x + i;
        if ( X >= 0 && X < Width && Y >= 0 && Y < Width ) {
          Dest[row] += Rows[row*9+k] * Arg[ startIndex ];
        }
        startIndex++;
        ++k;
      }
      startIndex += Width - 3;
    }
  }
}

template <typename RealType>
void GPU2DQuadraticGridMatrix<RealType>::apply ( const GPUVector<RealType> &Arg, GPUVector<RealType> &Dest ) const {
  //! \todo Make the block size variable.
#if 1
  cuda::GridBlockConfig2D gridConf( 16, 16, _gridWidth, _gridWidth );
  apply2DQuadraticGridMatrix <<<gridConf.dimGrid, gridConf.dimBlock, (gridConf.dimBlock.x+2) * (gridConf.dimBlock.y+2) * sizeof (RealType) >>>( this->getDataPointer(), Arg.getDataPointer(), Dest.getDataPointer(), _numRowAndCols, _gridWidth );
#else
  cuda::GridBlockConfig2D gridConf( 512, 1, _numRowAndCols, 1 );
  apply2DQuadraticGridMatrixSimple <<<gridConf.dimGrid, gridConf.dimBlock>>>( this->getDataPointer(), Arg.getDataPointer(), Dest.getDataPointer(), _numRowAndCols, _gridWidth );
#endif
}

template class GPU2DQuadraticGridMatrix<double>;
template class GPU2DQuadraticGridMatrix<float>;

// The CUSPARSE library needs CUDA 3.2 or later.
#if CUDA_VERSION >= 3020

#include <hipsparse.h>

CUDASparseHandle::CUDASparseHandle ( )
  : _pHandle ( new hipsparseHandle_t ) {
  if ( hipsparseCreate ( static_cast<hipsparseHandle_t*>( _pHandle ) ) != HIPSPARSE_STATUS_SUCCESS ) {
    fprintf( stderr, "Error while initializing the CUSPARSE library.\n" );
    abort();
  }
  else
    fprintf( stderr, "Successfully initialized the CUSPARSE library.\n" );
}

CUDASparseHandle::~CUDASparseHandle ( ) {
  hipsparseHandle_t *pHandle = static_cast<hipsparseHandle_t*>( _pHandle );
  if ( hipsparseDestroy ( *pHandle ) != HIPSPARSE_STATUS_SUCCESS )
    fprintf( stderr, "Error while deinitializing the CUSPARSE library.\n" );

  delete pHandle;
}

template <typename RealType>
GPUCSRMatrix<RealType>::GPUCSRMatrix ( const int NumRows,
                                       const int NumCols,
                                       const int NumEntries,
                                       const RealType *Values,
                                       const int *RowStartingIndices,
                                       const int *ColumnIndices,
                                       const CUDASparseHandle &SparseHandle )
  : _values ( NumEntries ),
    _rowStartingIndices ( NumRows + 1 ),
    _columnIndices ( NumEntries ),
    _numRows ( NumRows ),
    _numCols ( NumCols ),
    _pMatDescr ( new hipsparseMatDescr_t ),
    _sparseHandle ( SparseHandle ) {

  _values.copyFromHost ( Values );
  _rowStartingIndices.copyFromHost ( RowStartingIndices );
  _columnIndices.copyFromHost ( ColumnIndices );

  hipsparseMatDescr_t &descr = *(static_cast<hipsparseMatDescr_t *> ( _pMatDescr ));
  if ( hipsparseCreateMatDescr(&descr) != HIPSPARSE_STATUS_SUCCESS ) {
    fprintf( stderr, "Error calling hipsparseCreateMatDescr!\n" );
    abort();
  }

  hipsparseSetMatType ( descr, HIPSPARSE_MATRIX_TYPE_GENERAL );
  hipsparseSetMatIndexBase( descr, HIPSPARSE_INDEX_BASE_ZERO );
}

template <typename RealType>
GPUCSRMatrix<RealType>::~GPUCSRMatrix ( ) {
  hipsparseMatDescr_t *pMatDescr = static_cast<hipsparseMatDescr_t*>( _pMatDescr );
  if ( hipsparseDestroyMatDescr ( *pMatDescr ) != HIPSPARSE_STATUS_SUCCESS )
    fprintf( stderr, "Error calling hipsparseDestroyMatDescr!\n" );

  delete ( pMatDescr );
}

template <typename RealType>
void GPUCSRMatrix<RealType>::apply ( const GPUVector<RealType> &Arg, GPUVector<RealType> &Dest ) const {
  hipsparseScsrmv( *static_cast<const hipsparseHandle_t*>( _sparseHandle.getHandlePointer() ), HIPSPARSE_OPERATION_NON_TRANSPOSE,
                  _numRows, _numCols, 1, *static_cast<const hipsparseMatDescr_t*>( _pMatDescr ),
                  _values.getDataPointer(), _rowStartingIndices.getDataPointer(), _columnIndices.getDataPointer(),
                  Arg.getDataPointer(), 0, Dest.getDataPointer() );
}

template class GPUCSRMatrix<float>;

#endif 

}
